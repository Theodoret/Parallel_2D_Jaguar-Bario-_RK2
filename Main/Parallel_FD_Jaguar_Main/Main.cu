#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <cmath>

#define SizeX 1024
#define SizeY 1024

/* run this program using the console pauser or add your own getch, system("pause") or input loop */

// Non-CUDA stuffs
void show(double* x);
void writeText(double *u);
void readmatrix(double* a, const char* filename);

// CUDA stuffs
//  get k value
__global__
void getK(double *u1, double *v1, double *ku, double *kv) {
	// Working area
	int Nx = blockIdx.x * blockDim.x + threadIdx.x;
	int Ny = blockIdx.y * blockDim.y + threadIdx.y;
	int i = Nx + Ny * blockDim.x * gridDim.x;

	double dx;
	dx = 1.0;

	// Initialize hyperparameter
	double a = 0.45 * 6, b = 6, alp = 0.899, bet = -0.91, gam = -alp, r2 = 2.0, r3 = 3.5;

	double Ztop, Zleft, Zbottom, Zright, Zcenter;
	double deltau, deltav;

	//laplacian
	if (i > SizeY && i < SizeX * SizeY - SizeY && i % SizeY != 0 && i % SizeY != SizeY - 1) {
		Ztop = u1[i - SizeY];
		Zleft = u1[i - 1];
		Zbottom = u1[i + SizeY];
		Zright = u1[i + 1];
		Zcenter = u1[i];
		deltau = (Ztop + Zleft + Zbottom + Zright - 4 * Zcenter) / pow(dx, 2);
		//printf("\n deltau ke-%d = %f", i, deltau);
		//printf("\n u ke-%d = %f", i, dx);

		Ztop = v1[i - SizeY];
		Zleft = v1[i - 1];
		Zbottom = v1[i + SizeY];
		Zright = v1[i + 1];
		Zcenter = v1[i];
		deltav = (Ztop + Zleft + Zbottom + Zright - 4 * Zcenter) / pow(dx, 2);
	} else {
		deltau = u1[i];
		deltav = v1[i];
	}

	// Update
	ku[i] = (a * deltau + alp * u1[i] + v1[i] - r2 * u1[i] * v1[i]
		- alp * r3 * u1[i] * v1[i] * v1[i]);
	kv[i] = (b * deltav + gam * u1[i] + bet * v1[i] + r2 * u1[i] * v1[i]
		+ alp * r3 * u1[i] * v1[i] * v1[i]);
	//printf("\n ku ke-%d = %f", i, ku[i]);
}

__global__
void Neumann(double *u1, double *v1) {
	int Nx = blockIdx.x * blockDim.x + threadIdx.x;
	int Ny = blockIdx.y * blockDim.y + threadIdx.y;
	int i = Nx + Ny * blockDim.x * gridDim.x;

	if (i < SizeY) {
		u1[i] = u1[i + SizeY];
		v1[i] = v1[i + SizeY];
	}
	else if (i > SizeX*SizeY - SizeY) {
		u1[i] = u1[i - SizeY];
		v1[i] = v1[i - SizeY];
	}
	else if (i % SizeY == 0) {
		u1[i] = u1[i + 1];
		v1[i] = v1[i + 1];
	}
	else if (i % SizeY == SizeY - 1) {
		u1[i] = u1[i - 1];
		v1[i] = v1[i - 1];
	}
}

// Update U and V value
__global__
void update(double dt, double *k1u, double *k1v, double *u, double *v, double *u1, double *v1, double *ku, double *kv) {
	int Nx = blockIdx.x * blockDim.x + threadIdx.x;
	int Ny = blockIdx.y * blockDim.y + threadIdx.y;
	int i = Nx + Ny * blockDim.x * gridDim.x;
	
	u1[i] = u[i] + dt * (ku[i] + k1u[i]);
	v1[i] = v[i] + dt * (kv[i] + k1v[i]);
}

// Main Program
int main(int argc, char *argv[]) {
	// Index
	int i, j, k;
	double dx, dy;
	dx = (double) 2 / (SizeX - 1);
	dy = (double) 2 / (SizeY - 1);

	// 	Matrix size of x, y, and u with Nx as size of x and Ny as size of y
	int Nx = SizeX, Ny = SizeY;
	int elem = Nx*Ny;
	
	dim3 block(32 / 4, 32 / 4);
	dim3 grid((SizeY + block.x - 1) / block.x, (SizeX + block.y - 1) / block.y);

	// Time
	//double dt = 0.000005;
	double dt = 0.01;

	// Intialize array matrix of x, y, u, v, k, k1, and Area
	double *u, *v, *u1, *v1, *ku, *kv, *k1u, *k1v;
	double *d_u, *d_v, *d_u1, *d_v1, * d_ku, * d_kv, * d_k1u, * d_k1v;

	u = (double*)malloc(elem*sizeof(double));
	v = (double*)malloc(elem*sizeof(double));
	u1 = (double*)malloc(elem*sizeof(double));
	v1 = (double*)malloc(elem*sizeof(double));
	ku = (double*)malloc(elem * sizeof(double));
	kv = (double*)malloc(elem * sizeof(double));
	k1u = (double*)malloc(elem * sizeof(double));
	k1v = (double*)malloc(elem * sizeof(double));
	
	hipMalloc(&d_u, elem*sizeof(double));
	hipMalloc(&d_v, elem*sizeof(double));
	hipMalloc(&d_u1, elem*sizeof(double));
	hipMalloc(&d_v1, elem*sizeof(double));
	hipMalloc(&d_ku, elem * sizeof(double));
	hipMalloc(&d_kv, elem * sizeof(double));
	hipMalloc(&d_k1u, elem * sizeof(double));
	hipMalloc(&d_k1v, elem * sizeof(double));

	for (i = 0;i < elem;i++) {
		u[i] = rand() % 2;
		v[i] = rand() % 2;
	}

	//readmatrix(u, "u.txt");
	//readmatrix(v, "v.txt");

	// Send initialized values to Device
	hipMemcpy(d_u, u, elem * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_v, v, elem * sizeof(double), hipMemcpyHostToDevice);

	// Another magic batch!
	hipMemset(d_ku, 0, elem * sizeof(double));
	hipMemset(d_kv, 0, elem * sizeof(double));

	hipMemcpy(d_u1, d_u, elem * sizeof(double), hipMemcpyHostToHost);
	hipMemcpy(d_v1, d_v, elem * sizeof(double), hipMemcpyHostToHost);

	// Vibe Check
	//hipMemcpy(u1, d_u1, elem * sizeof(double), hipMemcpyDeviceToHost);
	//hipMemcpy(v1, d_v1, elem * sizeof(double), hipMemcpyDeviceToHost);
	//hipMemcpy(ku, d_ku, elem * sizeof(double), hipMemcpyDeviceToHost);
	//hipMemcpy(kv, d_kv, elem * sizeof(double), hipMemcpyDeviceToHost);

	//hipMemcpy(u, d_u, elem * sizeof(double), hipMemcpyDeviceToHost);
	//printf("\n\n\n U ");
	//show(u);

	//hipMemcpy(v, d_v, elem * sizeof(double), hipMemcpyDeviceToHost);
	//printf("\n\n\n V ");
	//show(v);

	//hipMemcpy(x, d_x, elem * sizeof(double), hipMemcpyDeviceToHost);
	//printf("\n\n\n x ");
	//show(x);

	//hipMemcpy(y, d_y, elem * sizeof(double), hipMemcpyDeviceToHost);
	//printf("\n\n\n y ");
	//show(y);

	// Record Time
	float elapsedTime = 0;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipSetDevice(0);
	hipEventRecord(start, 0);

	//double save[30000 / 3000][SizeX * SizeY];
	//i = 0;

	for (k = 0; k < 30000; k++) {
		printf("\n %d", k);
		// zero-ing temp
		hipMemset(d_k1u, 0, elem * sizeof(double));
		hipMemset(d_k1v, 0, elem * sizeof(double));

		// Copy U and V to U1 and V1
		hipMemcpy(d_u1, d_u, elem * sizeof(double), hipMemcpyDeviceToDevice);
		hipMemcpy(d_v1, d_v, elem * sizeof(double), hipMemcpyDeviceToDevice);



		// =============================================================================
		// K1 Stuffs
		// get K1

		getK << <grid, block >> > (d_u1, d_v1,d_ku, d_kv);

		// Update U1 and V1 first
		update << <grid, block >> > (dt / 1.0, d_k1u, d_k1v, d_u, d_v, d_u1, d_v1, d_ku, d_kv);

		// Neumann
		Neumann << <grid, block >> > (d_u1, d_v1);

		// Vibe Check
		//hipMemcpy(u1, d_u1, elem * sizeof(double), hipMemcpyDeviceToHost);
		//hipMemcpy(v1, d_v1, elem * sizeof(double), hipMemcpyDeviceToHost);
		//hipMemcpy(ku, d_ku, elem * sizeof(double), hipMemcpyDeviceToHost);
		//hipMemcpy(kv, d_kv, elem * sizeof(double), hipMemcpyDeviceToHost);

		//printf("\n U1 - %d", k);
		//show(u1);
		//printf("\n V1 - %d", k);
		//show(v1);
		//printf("\n K1U - %d", k);
		//show(ku);
		//printf("\n K1V - %d", k);
		//show(kv);
		// =============================================================================



		// =============================================================================
		// K2 Stuffs
		// allow move  ku value to k1u so ku can be come k2u (and so with kv)
		hipMemcpy(d_k1u, d_ku, elem * sizeof(double), hipMemcpyDeviceToDevice);
		hipMemcpy(d_k1v, d_kv, elem * sizeof(double), hipMemcpyDeviceToDevice);

		// get K2
		//hipMemcpy(d_Area, Area, elem * sizeof(double), hipMemcpyHostToDevice);

		getK << <grid, block >> > (d_u1, d_v1, d_ku, d_kv);

		//hipMemcpy(ku, d_ku, elem * sizeof(double), hipMemcpyDeviceToHost);
		//hipMemcpy(kv, d_kv, elem * sizeof(double), hipMemcpyDeviceToHost);

		// Update final U and V as U1 and V1

		update << <grid, block >> > (dt / 2.0, d_k1u, d_k1v, d_u, d_v, d_u1, d_v1, d_ku, d_kv);


		// Neumann
		Neumann << <grid, block >> > (d_u1, d_v1);

		// Vibe Check
		//hipMemcpy(u1, d_u1, elem * sizeof(double), hipMemcpyDeviceToHost);
		//hipMemcpy(v1, d_v1, elem * sizeof(double), hipMemcpyDeviceToHost);
		//hipMemcpy(ku, d_ku, elem * sizeof(double), hipMemcpyDeviceToHost);
		//hipMemcpy(kv, d_kv, elem * sizeof(double), hipMemcpyDeviceToHost);

		//printf("\n U2 - %d", k);
		//show(u1);
		//printf("\n V2 - %d", k);
		//show(v1);
		//printf("\n K2U - %d", k);
		//show(ku);
		//printf("\n K2V - %d", k);
		//show(kv);
		// =============================================================================



		// Finishing
		// get U and V final result from U1 and V1
		hipMemcpy(d_u, d_u1, elem * sizeof(double), hipMemcpyDeviceToDevice);
		hipMemcpy(d_v, d_v1, elem * sizeof(double), hipMemcpyDeviceToDevice);
		/*if (k % 3000 == 0) {
			hipMemcpy(u, d_u, elem * sizeof(double), hipMemcpyDeviceToHost);
			for (j = 0;j < elem;j++) {
				save[i][j] = u[j];
			}
			i++;
		}*/
	}

	// CUDA synchronize and get time
	hipDeviceSynchronize();

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	hipEventElapsedTime(&elapsedTime, start, stop);

	// Vibe Check
	hipMemcpy(u, d_u, elem * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(v, d_v, elem * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(u1, d_u1, elem * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(v1, d_v1, elem * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(ku, d_ku, elem * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(kv, d_kv, elem * sizeof(double), hipMemcpyDeviceToHost);

	printf("\n\n\n U - Update");
	show(u);

	printf("\n\n\n V - Update");
	show(v);

	printf("\n\n\n Lama Proses GPU = %.4fms", elapsedTime);

	writeText(u);
	hipFree(d_u); hipFree(d_v);
	hipFree(d_u1); hipFree(d_v1);
	hipFree(d_ku); hipFree(d_kv);
	free(u); free(v);
	free(u1); free(v1);
	free(ku); free(kv);

	return 0;
}


// Non-CUDA stuffs
void show(double* x) {
	for (int i = 0; i < SizeX * SizeY; i++) {
		if (i % SizeY == 0) {
			printf("\n[%.4f", x[i]);
		}
		else if (i % SizeY == SizeY - 1) {
			printf(", %.4f];", x[i]);
		}
		else {
			printf(", %.4f", x[i]);
		}
	}
}

// create .txt matrix result
void writeText(double *u){
	int i, j;
	
	const char *filename = "Matrix_Result.txt";

    // open the file for writing
    FILE *fp = fopen(filename, "w");
    // write to the text file
    for (i = 0; i < SizeX*SizeY; i++){
    	if(i%SizeY == 0){
		   	fprintf(fp, "\n[%.4f",u[i]);
		}else if(i%SizeY == SizeY-1){
			fprintf(fp, ", %.4f];",u[i]);
		}else{
			fprintf(fp, ", %.4f",u[i]);
		}
	}

    // close the file
    fclose(fp);

    return;
}

void readmatrix(double* a, const char* filename)
{
	int i, j;

	FILE* pf;
	pf = fopen(filename, "r");
	if (pf == NULL)
		return;

	for (i = 0; i < SizeX * SizeY; i++)
	{
		for (j = 0; j < SizeY; ++j) {
			fscanf(pf, "%lf", &a[i * SizeY + j]);
		}
	}

	fclose(pf);
	return;
}